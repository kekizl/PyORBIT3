#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "MatrixVec_kernel.cuh"

__global__ void matrixVectorMul(float *deviceMatrix, float *deviceVector, float *deviceResult, int matrixRows, int matrixCols, int vectorSize) {
    int idx = threadIdx.x;
    float sum = 0.0f;
    for (int i = 0; i < matrixCols; ++i) {
        sum += deviceMatrix[idx * matrixCols + i] * deviceVector[i];
    }
    deviceResult[idx] = sum;
}

void MatrixVecRun(float *deviceMatrix, float *deviceVector, float *deviceResult, int matrixRows, int matrixCols, int vectorSize) {
    matrixVectorMul<<<1, vectorSize>>>(deviceMatrix, deviceVector, deviceResult, matrixRows, matrixCols, vectorSize);
    hipDeviceSynchronize();
}

void MatrixVecCopy(float *deviceResult, float *hostResult, int vectorSize) {
    hipMemcpy(hostResult, deviceResult, vectorSize * sizeof(float), hipMemcpyDeviceToHost);
}

void MatrixVecInit(float **deviceMatrix, float **deviceVector, float **deviceResult, int matrixRows, int matrixCols, int vectorSize) {
    hipMalloc((void **)deviceMatrix, matrixRows * matrixCols * sizeof(float));
    hipMalloc((void **)deviceVector, vectorSize * sizeof(float));
    hipMalloc((void **)deviceResult, vectorSize * sizeof(float));

    // Copy data from host to device
    float hostMatrix[matrixRows * matrixCols] = {1.0f, 2.0f, 3.0f, 4.0f};
    float hostVector[vectorSize] = {5.0f, 6.0f};
    hipMemcpy(*deviceMatrix, hostMatrix, matrixRows * matrixCols * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(*deviceVector, hostVector, vectorSize * sizeof(float), hipMemcpyHostToDevice);
}

void MatrixVecFree(float *deviceMatrix, float *deviceVector, float *deviceResult) {
    hipFree(deviceMatrix);
    hipFree(deviceVector);
    hipFree(deviceResult);
}

